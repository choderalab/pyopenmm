#include "hip/hip_runtime.h"
/**
 * Reduce the derivatives computed in the N^2 energy kernel, and compute all per-particle energy terms.
 */

extern "C" __global__ void computePerParticleEnergy(long long* __restrict__ forceBuffers, real* __restrict__ energyBuffer, const real4* __restrict__ posq
        PARAMETER_ARGUMENTS) {
    real energy = 0;
    for (unsigned int index = blockIdx.x*blockDim.x+threadIdx.x; index < NUM_ATOMS; index += blockDim.x*gridDim.x) {
        // Load the derivatives

        LOAD_DERIVATIVES

        // Now calculate the per-particle energy terms.

        real4 pos = posq[index];
        real3 force = make_real3(0, 0, 0);
        COMPUTE_ENERGY
    }
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy;
}
